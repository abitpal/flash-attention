#include "hip/hip_runtime.h"
#include "utils.h"
#include "flash-attention-kernel.h"


template <typename T>
__global__ 
void flash_attn_forward(const T* Q, const T* K, const T* V,  const flash_attn_forward_params<T>* faf_param) {
    /*
        Input: 
            Q : Pointer to Q^T with shape (batch, num_head, d_k, num_seq_q) - lives in contiguous memory
            K : Pointer to K^T with shape (batch, num_head, d_k, num_seq_k) - lives in contiguous memory
            V : Pointer to V^T with shape (batch, num_head, d_v, num_seq_k) - lives in contiguous memory
        Return: 
            None - Modifies O
    */
    int tidx = threadIdx.x; // index of the specific row, d_idx < d_k
    int tcount = blockDim.x; 
    int batch_idx = blockIdx.x, head_idx = blockIdx.y;  // batch and head index
    int n_head = gridDim.y; 

    const int b_c = faf_param->b_c, b_r = faf_param->b_r, t_c = faf_param->t_c, t_r = faf_param->t_r; 
    const int n_seq_k = faf_param->n_seq_k, n_seq_q = faf_param->n_seq_q; 

    extern __shared__ T sram []; 
    T* q_i = sram; 
    T* k_i = sram + br * d_k; 
    T* v_i = k_i + bc * d_k; 
    T* o_i = v_i + bc * d_v; 

    // https://siboehm.com/articles/22/CUDA-MMM Global Memory Coalescing

    for (int q_idx = 0; q_idx < t_r; q_idx++) {
        // load q_i
        T* q_tile = _get_item(Q, {{-1, batch_idx}, {n_head, head_idx}, {d_k, 0}, {n_seq_q, q_idx * b_r}}); 
        _load_tile(q_tile, q_i, d_k, br, tidx, tcount); // this is a warp-aware load that loads w/ memory coalescing
        // init o_i to 0
        _fill(o_i, br * d_v, 0.0f, tcount, tidx); 
        int queries_per_thread = (d_k + tcount - 1) / tcount; // not on sram --> storage on register
        T q_max[queries_per_thread], q_sum[queries_per_thread]; 
        _fill_single_threaded(q_max, queries_per_thread, -INFINITY); 
        _fill_single_threaded(q_sum, queries_per_thread, 0.0f); 
        for (int k_idx = 0; k_idx < t_c; k_idx++) {
            // load k into sram
            T* k_tile = _get_item(K, {{-1, batch_idx}, {n_head, head_idx}, {d_k, 0}, {n_seq_k, k_idx * b_c}}); 
            T* v_tile = _get_item(V, {{-1, batch_idx}, {n_head, head_idx}, {d_v, 0}, {n_seq_k, k_idx * b_c}}); 
            _load_tile(k_tile, k_i, d_k, bc, tidx, tcount); 
            _load_tile(v_tile, v_i, d_k, bc, tidx, tcount); 
            __syncthreads(); 
            // compute Q * K
            _matmul_softmax(q_i, k_i, v_i, o_i, br, bc, d_k, tcount, q_max, q_sum); 
        }
        // final update on o_i
        _softmax_cumdiv(o_i, br, d_v, q_sum, tidx, tcount); 
        __synchthreads(); 
    }
}
