#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // at the top
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <vector>
#include <random>
#include <cmath>
#include <cassert>
#include "flash_attention_kernel.h"
#include <cmath>

// Helper function to check CUDA errors
#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Helper function to initialize random data
template <typename T>
void init_random_data(T* data, int size, T scale = 1.0f) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> dis(0.0f, scale);
    
    for (int i = 0; i < size; i++) {
        data[i] = static_cast<T>(dis(gen));
    }
}

template <typename T>
void reference_attention(const T* Q, const T* K, const T* V, T* O_ref,
                        int batch_size, int num_heads, int seq_len_q, int seq_len_k, 
                        int d_k, int d_v) {
    
    for (int b = 0; b < batch_size; b++) {
        for (int h = 0; h < num_heads; h++) {
            // Compute attention scores: Q * K^T
            std::vector<T> scores(seq_len_q * seq_len_k, 0);
            
            for (int i = 0; i < seq_len_q; i++) {
                for (int j = 0; j < seq_len_k; j++) {
                    T dot_product = 0;
                    for (int k = 0; k < d_k; k++) {
                        // Updated indexing for shape (b, nh, seq_len, d_k)
                        int q_idx = b * num_heads * seq_len_q * d_k + h * seq_len_q * d_k + i * d_k + k;
                        int k_idx = b * num_heads * seq_len_k * d_k + h * seq_len_k * d_k + j * d_k + k;
                        dot_product += Q[q_idx] * K[k_idx];
                    }
                    scores[i * seq_len_k + j] = dot_product / sqrt(static_cast<T>(d_k));
                }
            }
            
            // Apply softmax
            for (int i = 0; i < seq_len_q; i++) {
                T max_val = scores[i * seq_len_k];
                for (int j = 1; j < seq_len_k; j++) {
                    max_val = std::max(max_val, scores[i * seq_len_k + j]);
                }
                
                T sum_exp = 0;
                for (int j = 0; j < seq_len_k; j++) {
                    scores[i * seq_len_k + j] = exp(scores[i * seq_len_k + j] - max_val);
                    sum_exp += scores[i * seq_len_k + j];
                }
                
                for (int j = 0; j < seq_len_k; j++) {
                    scores[i * seq_len_k + j] /= sum_exp;
                }
            }
            
            // Compute output: scores * V
            for (int i = 0; i < seq_len_q; i++) {
                for (int k = 0; k < d_v; k++) {
                    T output_val = 0;
                    for (int j = 0; j < seq_len_k; j++) {
                        // Updated indexing for shape (b, nh, seq_len, d_v)
                        int v_idx = b * num_heads * seq_len_k * d_v + h * seq_len_k * d_v + j * d_v + k;
                        output_val += scores[i * seq_len_k + j] * V[v_idx];
                    }
                    // Updated indexing for output shape (b, nh, seq_len, d_v)
                    int o_idx = b * num_heads * seq_len_q * d_v + h * seq_len_q * d_v + i * d_v + k;
                    O_ref[o_idx] = output_val;
                }
            }
        }
    }
}

// Function to compare results
template <typename T>
bool compare_results(const T* gpu_result, const T* cpu_result, int size, T tolerance = 1e-3) {
    T max_diff = 0;
    int max_diff_idx = 0;
    
    for (int i = 0; i < size; i++) {
        T diff = std::abs(gpu_result[i] - cpu_result[i]);
        if (diff > max_diff) {
            max_diff = diff;
            max_diff_idx = i;
        }
    }
    
    std::cout << "Max difference: " << max_diff << " at index " << max_diff_idx << std::endl;
    std::cout << "GPU value: " << gpu_result[max_diff_idx] << ", CPU value: " << cpu_result[max_diff_idx] << std::endl;
    
    return max_diff < tolerance;
}

int main() {

    int device = 0; 
    hipDeviceProp_t prop;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    std::cout << "Device: " << prop.name << std::endl;
    std::cout << "Shared memory per block: " << prop.sharedMemPerBlock << " bytes" << std::endl;

    int sram_size_limit = prop.sharedMemPerBlock / sizeof(float); 

    // Test parameters - 4, 8, 128, 64
    const int batch_size = 4;
    const int num_heads = 8;
    const int seq_len_q = 128; 
    // const int seq_len_q = 1024; 
    const int seq_len_k = seq_len_q; 
    const int d_k = 64;
    const int d_v = 64;
    const float scaling_factor = 1.0f/sqrtf(static_cast<float>(d_k)); 
    
    // Flash attention parameters
    // const int b_r = min(seq_len_q, (sram_size_limit / ((d_k + d_v) * 2)));  // block rows (query block size)
    // const int b_c = min(min(d_k, seq_len_k), (sram_size_limit / ((d_k + d_v) * 2)));  // block columns (key/value block size)
    const int offset = -16; 
    const int b_r = 48 - offset; 
    const int b_c = 48 + offset; 
    const int t_r = (seq_len_q + b_r - 1) / b_r;  // number of query tiles
    const int t_c = (seq_len_k + b_c - 1) / b_c;  // number of key tiles

    int sram_size = (b_r * d_k + b_c * d_k + b_c * d_v + b_r * d_v) * sizeof(float); // = (b_r + b_c) * (d_k + d_v)
    /*
    sram_size / (d_k + d_v) / 2
    */
    std::cout << "Shared memory size: " << sram_size << " bytes" << std::endl;
    assert((sram_size <= prop.sharedMemPerBlock)); 
    
    std::cout << "Test Parameters:" << std::endl;
    std::cout << "Batch size: " << batch_size << std::endl;
    std::cout << "Num heads: " << num_heads << std::endl;
    std::cout << "Seq len Q: " << seq_len_q << ", Seq len K: " << seq_len_k << std::endl;
    std::cout << "d_k: " << d_k << ", d_v: " << d_v << std::endl;
    std::cout << "Block size (b_r, b_c): (" << b_r << ", " << b_c << ")" << std::endl;
    std::cout << "Tiles (t_r, t_c): (" << t_r << ", " << t_c << ")" << std::endl;
    
    // Calculate sizes
    const int q_size = batch_size * num_heads * d_k * seq_len_q;
    const int k_size = batch_size * num_heads * d_k * seq_len_k;
    const int v_size = batch_size * num_heads * d_v * seq_len_k;
    const int o_size = batch_size * num_heads * d_v * seq_len_q;
    
    // Allocate host memory
    std::vector<float> h_Q(q_size);
    std::vector<float> h_K(k_size);
    std::vector<float> h_V(v_size);
    std::vector<float> h_O(o_size);
    std::vector<float> h_O_ref(o_size);
    
    // Initialize input data
    std::cout << "Initializing input data..." << std::endl;
    init_random_data(h_Q.data(), q_size, 0.1f);
    init_random_data(h_K.data(), k_size, 0.1f);
    init_random_data(h_V.data(), v_size, 0.1f);
    
    // Allocate device memory
    float *d_Q, *d_K, *d_V, *d_O; 
    flash_attn_forward_params *d_params;
    
    CUDA_CHECK(hipMalloc(&d_Q, q_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_K, k_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_V, v_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_O, o_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_params, sizeof(flash_attn_forward_params)));
    
    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_Q, h_Q.data(), q_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_K, h_K.data(), k_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_V, h_V.data(), v_size * sizeof(float), hipMemcpyHostToDevice));
            
    // Launch kernel
    std::cout << "Launching Flash Attention kernel..." << std::endl;
    
    dim3 grid(batch_size, num_heads);
    int block_x = 8; 
    int block_y = b_r; 
    int block_z = 1; 
    dim3 block(block_x, block_y, block_z); // Ensure block size doesn't exceed max

    // std::cout << "Block dim: " << 1024 / b_r << ' ' << b_r << '\n'; 

    // std::cout << "Key:\n"; 
    // for (int i = 0; i < d_k; i++) {
    //     for (int j = 0; j < seq_len_q; j++) {
    //         std::cout << h_K[i * seq_len_q + j] << ' '; 
    //     }
    //     std::cout << '\n'; 
    // }
    // std::cout << '\n'; 
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    hipProfilerStart();  // Begin nsys profiling window

    CUDA_CHECK(hipEventRecord(start));
    flash_attn_forward<<<grid, block, sram_size>>>(d_Q, d_K, d_V, d_O, b_c, b_r, t_c, t_r, seq_len_k, seq_len_q, d_k, scaling_factor);
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipDeviceSynchronize());  // Wait for kernel to finish
    hipProfilerStop();  // End nsys profiling window

    CUDA_CHECK(hipGetLastError());  // Check for errors
        
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;
    
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_O.data(), d_O, o_size * sizeof(float), hipMemcpyDeviceToHost));
    
    // Compute reference result
    // std::cout << "Computing reference result..." << std::endl;
    // reference_attention(h_Q.data(), h_K.data(), h_V.data(), h_O_ref.data(),
    //                    batch_size, num_heads, seq_len_q, seq_len_k, d_k, d_v);
    
    // // Compare results
    // std::cout << "Comparing results..." << std::endl;
    // bool passed = compare_results(h_O.data(), h_O_ref.data(), o_size, 1e-2f);
    
    // if (passed) {
    //     std::cout << "Test PASSED!" << std::endl;
    // } else {
    //     std::cout << "Test FAILED!" << std::endl;
    // }
    
    // // Print some sample values for inspection
    // std::cout << "\nSample values comparison:" << std::endl;
    // for (int i = 0; i < std::min(10, o_size); i++) {
    //     std::cout << "Index " << i << ": GPU=" << h_O[i] << ", CPU=" << h_O_ref[i] << std::endl;
    // }

    // print O

    // for (int i = 0; i < d_k; i++) {
    //     for (int j = 0; j < seq_len_q; j++) {
    //         printf("%.5f=%.5f ", h_O[i * seq_len_q + j], h_O_ref[i * seq_len_q + j]); 
    //     }
    //     std::cout << '\n'; 
    // }
    
    // Cleanup
    CUDA_CHECK(hipFree(d_Q));
    CUDA_CHECK(hipFree(d_K));
    CUDA_CHECK(hipFree(d_V));
    CUDA_CHECK(hipFree(d_O));
    CUDA_CHECK(hipFree(d_params));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    
    return 0; 
}