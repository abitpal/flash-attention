#include "hip/hip_runtime.h"
#include "utils.h"
#include "flash_attention_kernel.h"
#include "consts.h"
#include <assert.h>
#include <cstdio> 

__global__ 
void flash_attn_forward(float* Q, float* K, float* V, float* O, flash_attn_forward_params* faf_param) {
    int tidx = threadIdx.x; // index of the specific row, d_idx < d_k
    int tcount = blockDim.x; 
    int batch_idx = blockIdx.x, head_idx = blockIdx.y;  // batch and head index
    int n_head = gridDim.y; 

    const int b_c = faf_param->b_c, b_r = faf_param->b_r, t_c = faf_param->t_c, t_r = faf_param->t_r; 
    const int d_k = faf_param->d_k, d_v = faf_param->d_v; 
    const int n_seq_k = faf_param->n_seq_k, n_seq_q = faf_param->n_seq_q; 
    const float scaling_factor = faf_param->scaling_factor; 

    int queries_per_thread = (b_r + tcount - 1) / tcount; // not on sram --> storage on register

    extern __shared__ float sram []; 
    float* q_i = sram; 
    float* k_i = sram + b_r * d_k; 
    float* v_i = k_i + b_c * d_k; 
    float* o_i = v_i + b_c * d_v; 

    // printf("queries per thread: %d, d_k: %d tcount: %d\n", queries_per_thread, d_k, tcount); 
    assert((queries_per_thread <= max_queries_per_thread)); 

    // https://siboehm.com/articles/22/CUDA-MMM Global Memory Coalescing

    for (int q_idx = 0; q_idx < t_r; q_idx++) {
        // load q_i
        int loc_q_tile[4][2] = {{-1, batch_idx},{n_head, head_idx}, {d_k, 0}, {n_seq_q, 0}};
        float* q_tile = _get_item(Q, loc_q_tile, 4); 
        // printf("start col: %d, end col: %d\n", q_idx * b_r, (q_idx + 1) * b_r - 1); 
        _load_tile(q_tile, q_i, n_seq_q, 0, d_k - 1, q_idx * b_r, (q_idx + 1) * b_r - 1, tidx, tcount); // this is a warp-aware load that loads w/ memory coalescing
        // init o_i to 0
        _fill(o_i, b_r * d_v, 0.0f, tcount, tidx); 
        float q_max[max_queries_per_thread], q_sum[max_queries_per_thread]; 
        _fill_single_threaded(q_max, queries_per_thread, -INFINITY); 
        _fill_single_threaded(q_sum, queries_per_thread, 0.0f); 

        for (int k_idx = 0; k_idx < t_c; k_idx++) {
            // load k into sram
            int loc_k_tile[4][2] = {{-1, batch_idx}, {n_head, head_idx}, {d_k, 0}, {n_seq_k, 0}}; 
            int loc_v_tile[4][2] =  {{-1, batch_idx}, {n_head, head_idx}, {d_v, 0}, {n_seq_k, 0}}; 
            float* k_tile = _get_item(K, loc_k_tile, 4); 
            float* v_tile = _get_item(V, loc_v_tile, 4); 
            _load_tile(k_tile, k_i, n_seq_k, 0, d_k - 1, k_idx * b_c, (k_idx + 1) * b_c - 1, tidx, tcount);
            _load_tile(v_tile, v_i, n_seq_k, 0, d_v - 1, k_idx * b_c, (k_idx + 1) * b_c - 1, tidx, tcount); 
            __syncthreads(); 
            // compute (Q * K) * V 
            _matmul_softmax(q_i, k_i, v_i, o_i, b_r, b_c, d_k, d_v, tcount, tidx, q_max, q_sum, 1.0f); 
        }
        // final update on o_i w/ the softmax division
        _softmax_cumdiv(o_i, b_r, d_v, q_sum, tidx, tcount); 

        // write back O to HBM
        for (int i = 0; i < d_v; i++) {
            for (int j = tidx; j < b_r; j += tcount) {
                int loc_O[4][2] = {{-1, batch_idx}, {n_head, head_idx}, {d_v, i}, {n_seq_q, j + q_idx * b_r}}; 
                float* O_idx = _get_item(O, loc_O, 4); 
                *O_idx = o_i[i * b_r + j]; 
            }
        }
        __syncthreads(); 
    }
}

